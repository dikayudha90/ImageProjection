#include "hip/hip_runtime.h"
#include "CUDA_RANSAC_Homography.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <numeric>
#include <omp.h>
#include <assert.h>
#include "CUDA_SVD.cu"

static const int NTHREADS = 512; // threads per block

#define SQ(x) (x)*(x)

static void CheckCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

__device__ int CalcHomography(const Point2Df src[4], const Point2Df dst[4], float ret_H[9])
{
    // This version does not normalised the input data, which is contrary to what Multiple View Geometry says.
    // I included it to see what happens when you don't do this step.

    float X[M*N]; // M,N #define inCUDA_SVD.cu

    for(int i=0; i < 4; i++) {
        float srcx = src[i].x;
        float srcy = src[i].y;
        float dstx = dst[i].x;
        float dsty = dst[i].y;

        int y1 = (i*2 + 0)*N;
        int y2 = (i*2 + 1)*N;

        // First row
        X[y1+0] = 0.f;
        X[y1+1] = 0.f;
        X[y1+2] = 0.f;

        X[y1+3] = -srcx;
        X[y1+4] = -srcy;
        X[y1+5] = -1.f;

        X[y1+6] = dsty*srcx;
        X[y1+7] = dsty*srcy;
        X[y1+8] = dsty;

        // Second row
        X[y2+0] = srcx;
        X[y2+1] = srcy;
        X[y2+2] = 1.f;

        X[y2+3] = 0.f;
        X[y2+4] = 0.f;
        X[y2+5] = 0.f;

        X[y2+6] = -dstx*srcx;
        X[y2+7] = -dstx*srcy;
        X[y2+8] = -dstx;
    }

    // Fill the last row
    float srcx = src[3].x;
    float srcy = src[3].y;
    float dstx = dst[3].x;
    float dsty = dst[3].y;

    int y = 8*N;
    X[y+0] = -dsty*srcx;
    X[y+1] = -dsty*srcy;
    X[y+2] = -dsty;

    X[y+3] = dstx*srcx;
    X[y+4] = dstx*srcy;
    X[y+5] = dstx;

    X[y+6] = 0;
    X[y+7] = 0;
    X[y+8] = 0;

    float w[N];
    float v[N*N];

    int ret = dsvd(X, M, N, w, v);

    if(ret == 1) {
        // Sort
        float smallest = w[0];
        int col = 0;

        for(int i=1; i < N; i++) {
            if(w[i] < smallest) {
                smallest = w[i];
                col = i;
            }
        }

        ret_H[0] = v[0*N + col];
        ret_H[1] = v[1*N + col];
        ret_H[2] = v[2*N + col];
        ret_H[3] = v[3*N + col];
        ret_H[4] = v[4*N + col];
        ret_H[5] = v[5*N + col];
        ret_H[6] = v[6*N + col];
        ret_H[7] = v[7*N + col];
        ret_H[8] = v[8*N + col];
    }

    return ret;
}

__device__ int CalcHomography2(const Point2Df src[4], const Point2Df dst[4], float ret_H[9])
{
    // This version normalises the data before processing as recommended in the book Multiple View Geometry
    // But for some reason it can perform worse than the unnormalised version (less inliers detected).
    // Something to do with using floats only?

    float X[M*N]; // M,N #define inCUDA_SVD.cu

    // Normalise the data
    Point2Df src_mean, dst_mean;
    float src_var = 0.0f;
    float dst_var = 0.0f;

    src_mean.x = (src[0].x + src[1].x + src[2].x + src[3].x)*0.25f;
    src_mean.y = (src[0].y + src[1].y + src[2].y + src[3].y)*0.25f;

    dst_mean.x = (dst[0].x + dst[1].x + dst[2].x + dst[3].x)*0.25f;
    dst_mean.y = (dst[0].y + dst[1].y + dst[2].y + dst[3].y)*0.25f;

    for(int i=0; i < 4; i++) {
        src_var += SQ(src[i].x - src_mean.x) + SQ(src[i].y - src_mean.y);
        dst_var += SQ(dst[i].x - dst_mean.x) + SQ(dst[i].y - dst_mean.y);
    }

    src_var *= 0.25f;
    dst_var *= 0.25f;

    float src_scale = sqrt(2.0f) / sqrt(src_var);
    float dst_scale = sqrt(2.0f) / sqrt(dst_var);

    for(int i=0; i < 4; i++) {
        float srcx = (src[i].x - src_mean.x)*src_scale;
        float srcy = (src[i].y - src_mean.y)*src_scale;
        float dstx = (dst[i].x - dst_mean.x)*dst_scale;
        float dsty = (dst[i].y - dst_mean.y)*dst_scale;

        int y1 = (i*2 + 0)*N;
        int y2 = (i*2 + 1)*N;

        // First row
        X[y1+0] = 0.0f;
        X[y1+1] = 0.0f;
        X[y1+2] = 0.0f;

        X[y1+3] = -srcx;
        X[y1+4] = -srcy;
        X[y1+5] = -1.f;

        X[y1+6] = dsty*srcx;
        X[y1+7] = dsty*srcy;
        X[y1+8] = dsty;

        // Second row
        X[y2+0] = srcx;
        X[y2+1] = srcy;
        X[y2+2] = 1.0f;

        X[y2+3] = 0.0f;
        X[y2+4] = 0.0f;
        X[y2+5] = 0.0f;

        X[y2+6] = -dstx*srcx;
        X[y2+7] = -dstx*srcy;
        X[y2+8] = -dstx;
    }

    // Fill the last row
    float srcx = src[3].x;
    float srcy = src[3].y;
    float dstx = dst[3].x;
    float dsty = dst[3].y;

    int y = 8*N;
    X[y+0] = -dsty*srcx;
    X[y+1] = -dsty*srcy;
    X[y+2] = -dsty;

    X[y+3] = dstx*srcx;
    X[y+4] = dstx*srcy;
    X[y+5] = dstx;

    X[y+6] = 0;
    X[y+7] = 0;
    X[y+8] = 0;

    float w[N];
    float v[N*N];

    int ret = dsvd(X, M, N, w, v);

    if(ret == 1) {
        // Sort
        float smallest = w[0];
        int col = 0;

        for(int i=1; i < N; i++) {
            if(w[i] < smallest) {
                smallest = w[i];
                col = i;
            }
        }

        float H[9];

        H[0] = v[0*N + col];
        H[1] = v[1*N + col];
        H[2] = v[2*N + col];
        H[3] = v[3*N + col];
        H[4] = v[4*N + col];
        H[5] = v[5*N + col];
        H[6] = v[6*N + col];
        H[7] = v[7*N + col];
        H[8] = v[8*N + col];

        // Undo the transformation using inv(dst_transform) * H * src_transform
        // Matrix operation expanded out, thanks to wxMaxima software
        float s1 = dst_scale;
        float s2 = src_scale;
        float tx1 = dst_mean.x;
        float ty1 = dst_mean.y;
        float tx2 = src_mean.x;
        float ty2 = src_mean.y;

        ret_H[0] = s2*tx1*H[6] + s2*H[0]/s1;
        ret_H[1] = s2*tx1*H[7] + s2*H[1]/s1;
        ret_H[2] = tx1*(H[8] - s2*ty2*H[7] - s2*tx2*H[6]) + (H[2] - s2*ty2*H[1] - s2*tx2*H[0])/s1;
        ret_H[3] = s2*ty1*H[6] + s2*H[3]/s1;
        ret_H[4] = s2*ty1*H[7] + s2*H[4]/s1;
        ret_H[5] = ty1*(H[8] - s2*ty2*H[7] - s2*tx2*H[6]) + (H[5] - s2*ty2*H[4] - s2*tx2*H[3])/s1;
        ret_H[6] = s2*H[6];
        ret_H[7] = s2*H[7];
        ret_H[8] = H[8] - s2*ty2*H[7] - s2*tx2*H[6];
    }

    return ret;
}

__device__ int EvalHomography(const Point2Df *src, const Point2Df *dst, int pts_num, const float H[9], float inlier_threshold)
{
    int inliers = 0;

    for(int i=0; i < pts_num; i++) {
        float x = H[0]*src[i].x + H[1]*src[i].y + H[2];
        float y = H[3]*src[i].x + H[4]*src[i].y + H[5];
        float z = H[6]*src[i].x + H[7]*src[i].y + H[8];

        x /= z;
        y /= z;

        float dist_sq = (dst[i].x - x)*(dst[i].x- x) + (dst[i].y - y)*(dst[i].y - y);

        if(dist_sq < inlier_threshold) {
            inliers++;
        }
    }

    return inliers;
}

__global__ void RANSAC_Homography(const Point2Df *src, const Point2Df *dst,int pts_num, const int *rand_list, float inlier_threshold, int iterations, int *ret_inliers, float *ret_homography)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if(idx >= iterations) {
        return;
    }

    ret_inliers[idx] = 0;

    int rand_idx[4];
    Point2Df _src[4];
    Point2Df _dst[4];
    float *H = &ret_homography[idx*9];

    rand_idx[0] = rand_list[idx*4];
    rand_idx[1] = rand_list[idx*4+1];
    rand_idx[2] = rand_list[idx*4+2];
    rand_idx[3] = rand_list[idx*4+3];

    // Check for duplicates
    if(rand_idx[0] == rand_idx[1]) return;
    if(rand_idx[0] == rand_idx[2]) return;
    if(rand_idx[0] == rand_idx[3]) return;
    if(rand_idx[1] == rand_idx[2]) return;
    if(rand_idx[1] == rand_idx[3]) return;
    if(rand_idx[2] == rand_idx[3]) return;

    for(int i=0; i < 4; i++) {
        _src[i].x = src[rand_idx[i]].x;
        _src[i].y = src[rand_idx[i]].y;
        _dst[i].x = dst[rand_idx[i]].x;
        _dst[i].y = dst[rand_idx[i]].y;
    }

#ifdef NORMALISE_INPUT_POINTS
    int ret = CalcHomography2(_src, _dst, H);
#else
    int ret = CalcHomography(_src, _dst, H);
#endif

    ret_inliers[idx] = EvalHomography(src, dst, pts_num, H, inlier_threshold);
}

void CUDA_RANSAC_Homography(const vector <Point2Df> &src, const vector <Point2Df> &dst, const vector <float> &match_score,
                            float inlier_threshold, int iterations,
                            int *best_inliers, float *best_H, vector <char> *inlier_mask)
{
    assert(src.size() == dst.size());
    assert(match_score.size() == dst.size());

    int RANSAC_threshold = inlier_threshold*inlier_threshold;
    int threads = NTHREADS;
    int blocks = iterations/threads + ((iterations % threads)?1:0);

    Point2Df *gpu_src;
    Point2Df *gpu_dst;
    int *gpu_rand_list;
    int *gpu_ret_inliers;
    float *gpu_ret_H;
    vector <int> rand_list(iterations*4);
    vector <int> ret_inliers(iterations);
    vector <float> ret_H(iterations*9);

    hipMalloc((void**)&gpu_src, sizeof(Point2Df)*src.size());
    hipMalloc((void**)&gpu_dst, sizeof(Point2Df)*dst.size());
    hipMalloc((void**)&gpu_rand_list, sizeof(int)*iterations*4);
    hipMalloc((void**)&gpu_ret_inliers, sizeof(int)*iterations);
    hipMalloc((void**)&gpu_ret_H, sizeof(float)*iterations*9);
    CheckCUDAError("hipMalloc");

    hipMemcpy(gpu_src, &src[0], sizeof(Point2Df)*src.size(), hipMemcpyHostToDevice);
    hipMemcpy(gpu_dst, &dst[0], sizeof(Point2Df)*dst.size(), hipMemcpyHostToDevice);

    // Generate random numbers on host
    // Using a bias version when randomly selecting points
    // Point with better matching score have a highr chance of getting picked
    {
#ifdef BIAS_RANDOM_SELECTION
        vector <float> cummulative = match_score;
        double sum = accumulate(match_score.begin(), match_score.end(), 0.0);

        // Normalise the scores
        for(unsigned int i=0; i < cummulative.size(); i++) {
            cummulative[i] /= sum;
        }

        // Calc the cummulative distribution
        for(unsigned int i=1; i < cummulative.size(); i++) {
            cummulative[i] += cummulative[i-1];
        }

        for(unsigned int i=0; i < rand_list.size(); i++) {
            float x = rand()/(1.0 + RAND_MAX); // random between [0,1)

            // Binary search to find which index x lands on
            int min = 0;
            int max = src.size();
            int index = 0;

            while(true) {
                int mid = (min + max) / 2;

                if(min == max - 1) {
                    if(x < cummulative[min]) {
                        index = min;
                    }
                    else {
                        index = max;
                    }
                    break;
                }

                if(x > cummulative[mid]) {
                    min = mid;
                }
                else {
                    max = mid;
                }
            }

            rand_list[i] = index;
        }
#else
        for(unsigned int i=0; i < rand_list.size(); i++) {
            rand_list[i] = src.size() * (rand()/(1.0 + RAND_MAX));
        }
#endif
        hipMemcpy(gpu_rand_list, &rand_list[0], sizeof(int)*rand_list.size(), hipMemcpyHostToDevice);
        CheckCUDAError("hipMemcpy");
    }

    RANSAC_Homography<<<blocks, threads>>>(gpu_src, gpu_dst, src.size(), gpu_rand_list, RANSAC_threshold, iterations, gpu_ret_inliers, gpu_ret_H);
    hipDeviceSynchronize();
    CheckCUDAError("RANSAC_Homography");

    hipMemcpy(&ret_inliers[0], gpu_ret_inliers, sizeof(int)*ret_inliers.size(), hipMemcpyDeviceToHost);
    hipMemcpy(&ret_H[0], gpu_ret_H, sizeof(float)*ret_H.size(), hipMemcpyDeviceToHost);

    *best_inliers = 0;
    int best_idx = 0;

    for(int i=0; i < ret_inliers.size(); i++) {
        /*
        printf("ret %d: %d\n", i, ret_inliers[i]);

        for(int j=0; j< 9; j++) {
            printf("%.3f ",  ret_H[i*9+j]);
        }
        printf("\n");
        */
        if(ret_inliers[i] > *best_inliers) {
            *best_inliers = ret_inliers[i];
            best_idx = i;
        }
    }

    memcpy(best_H, &ret_H[best_idx*9], sizeof(float)*9);

    // Fill the mask
    vector <char> &_inlier_mask = *inlier_mask;
    _inlier_mask.resize(src.size(), 0);

    for(int i=0; i < src.size(); i++) {
        float x = best_H[0]*src[i].x + best_H[1]*src[i].y + best_H[2];
        float y = best_H[3]*src[i].x + best_H[4]*src[i].y + best_H[5];
        float z = best_H[6]*src[i].x + best_H[7]*src[i].y + best_H[8];

        x /= z;
        y /= z;

        float dist_sq = (dst[i].x - x)*(dst[i].x- x) + (dst[i].y - y)*(dst[i].y - y);

        if(dist_sq < RANSAC_threshold) {
            _inlier_mask[i] = 1;
        }
    }

    *best_inliers = accumulate(_inlier_mask.begin(), _inlier_mask.end(), 0);

    //printf("CUDA blocks/threads: %d %d\n", blocks, threads);

    hipFree(gpu_src);
    hipFree(gpu_dst);
    hipFree(gpu_rand_list);
    hipFree(gpu_ret_inliers);
    hipFree(gpu_ret_H);
}
